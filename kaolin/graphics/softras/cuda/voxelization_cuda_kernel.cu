#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
// #
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// #
//     http://www.apache.org/licenses/LICENSE-2.0
// #
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


// Soft Rasterizer (SoftRas)

// Copyright (c) 2017 Hiroharu Kato
// Copyright (c) 2018 Nikos Kolotouros
// Copyright (c) 2019 Shichen Liu

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// #
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// #
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <iostream>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

// for the older gpus atomicAdd with double arguments does not exist
#if  __CUDA_ARCH__ < 600 && defined(__CUDA_ARCH__)
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif



namespace{


template <typename scalar_t>
__global__ void voxelize_sub1_kernel(
        const scalar_t* __restrict__ faces,
        int32_t* voxels,
        int batch_size,
        int num_faces,
        int voxel_size) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * voxel_size * voxel_size) {
        return;
    }
    const int bs = batch_size;
    const int nf = num_faces;
    const int vs = voxel_size;

    int y = i % vs;
    int x = (i / vs) % vs;
    int bn = i / (vs * vs);
    //
    for (int fn = 0; fn < nf; fn++) {
        const scalar_t* face = &faces[(bn * nf + fn) * 9];
        scalar_t y1d = face[3] - face[0];
        scalar_t x1d = face[4] - face[1];
        scalar_t z1d = face[5] - face[2];
        scalar_t y2d = face[6] - face[0];
        scalar_t x2d = face[7] - face[1];
        scalar_t z2d = face[8] - face[2];
        scalar_t ypd = y - face[0];
        scalar_t xpd = x - face[1];
        scalar_t det = x1d * y2d - x2d * y1d;
        if (det == 0) continue;
        scalar_t t1 = (y2d * xpd - x2d * ypd) / det;
        scalar_t t2 = (-y1d * xpd + x1d * ypd) / det;
        if (t1 < 0) continue;
        if (t2 < 0) continue;
        if (1 < t1 + t2) continue;
        int zi = floor(t1 * z1d + t2 * z2d + face[2]);
        int yi, xi;
        yi = y;
        xi = x;
        if ((0 <= yi) && (yi < vs) && (0 <= xi) && (xi < vs) && (0 <= zi) && (zi < vs))
            voxels[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] = 1;
        yi = y - 1;
        xi = x;
        if ((0 <= yi) && (yi < vs) && (0 <= xi) && (xi < vs) && (0 <= zi) && (zi < vs))
            voxels[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] = 1;
        yi = y;
        xi = x - 1;
        if ((0 <= yi) && (yi < vs) && (0 <= xi) && (xi < vs) && (0 <= zi) && (zi < vs))
            voxels[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] = 1;
        yi = y - 1;
        xi = x - 1;
        if ((0 <= yi) && (yi < vs) && (0 <= xi) && (xi < vs) && (0 <= zi) && (zi < vs))
            voxels[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] = 1;
    }
}


template <typename scalar_t>
__global__ void voxelize_sub2_kernel(
        const scalar_t* __restrict__ faces,
        int32_t* voxels,
        int batch_size,
        int num_faces,
        int voxel_size) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * num_faces) {
        return;
    }
    const int bs = batch_size;
    const int nf = num_faces;
    const int vs = voxel_size;

    int fn = i % nf;
    int bn = i / nf;
    const scalar_t* face = &faces[(bn * nf + fn) * 9];
    for (int k = 0; k < 3; k++) {
        int yi = floor(face[3 * k + 0]);
        int xi = floor(face[3 * k + 1]);
        int zi = floor(face[3 * k + 2]);
        if ((0 <= yi) && (yi < vs) && (0 <= xi) && (xi < vs) && (0 <= zi) && (zi < vs)) {
            voxels[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] = 1;
        }
    }
}

template <typename scalar_t>
__global__ void voxelize_sub3_kernel(
        int32_t* voxels,
        int32_t* visible,
        int batch_size,
        int voxel_size) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * voxel_size * voxel_size * voxel_size) {
        return;
    }
    const int bs = batch_size;
    const int vs = voxel_size;

    int z = i % vs;
    int x = (i / vs) % vs;
    int y = (i / (vs * vs)) % vs;
    int bn = i / (vs * vs * vs);
    int pn = i;
    if ((y == 0) || (y == vs - 1) || (x == 0) || (x == vs - 1) || (z == 0) || (z == vs - 1)) {
        if (voxels[pn] == 0) visible[pn] = 1;
    }
}

template <typename scalar_t>
__global__ void voxelize_sub4_kernel(
        int32_t* voxels,
        int32_t* visible,
        int batch_size,
        int voxel_size) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size * voxel_size * voxel_size * voxel_size) {
        return;
    }
    const int bs = batch_size;
    const int vs = voxel_size;

    int z = i % vs;
    int x = (i / vs) % vs;
    int y = (i / (vs * vs)) % vs;
    int bn = i / (vs * vs * vs);
    int pn = i;
    if ((y == 0) || (y == vs - 1) || (x == 0) || (x == vs - 1) || (z == 0) || (z == vs - 1)) return;
    if (voxels[pn] == 0 && visible[pn] == 0) {
        int yi, xi, zi;
        yi = y - 1;
        xi = x;
        zi = z;
        if (visible[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] != 0) visible[pn] = 1;
        yi = y + 1;
        xi = x;
        zi = z;
        if (visible[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] != 0) visible[pn] = 1;
        yi = y;
        xi = x - 1;
        zi = z;
        if (visible[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] != 0) visible[pn] = 1;
        yi = y;
        xi = x + 1;
        zi = z;
        if (visible[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] != 0) visible[pn] = 1;
        yi = y;
        xi = x;
        zi = z - 1;
        if (visible[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] != 0) visible[pn] = 1;
        yi = y;
        xi = x;
        zi = z + 1;
        if (visible[bn * vs * vs * vs + yi * vs * vs + xi * vs + zi] != 0) visible[pn] = 1;
    }
}

}



std::vector<at::Tensor> voxelize_sub1_cuda(
        at::Tensor faces,
        at::Tensor voxels) {

    const auto batch_size = faces.size(0);
    const auto num_faces = faces.size(1);
    const auto voxel_size = voxels.size(1);
    const int threads = 512;
    const dim3 blocks ((batch_size * voxel_size * voxel_size - 1) / threads +1);

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "voxelize_sub1_cuda", ([&] {
      voxelize_sub1_kernel<scalar_t><<<blocks, threads>>>(
          faces.data<scalar_t>(),
          voxels.data<int32_t>(),
          batch_size,
          num_faces,
          voxel_size);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in voxelize_sub1_kernel: %s\n", hipGetErrorString(err));

    return {voxels};
}



std::vector<at::Tensor> voxelize_sub2_cuda(
        at::Tensor faces,
        at::Tensor voxels) {

    const auto batch_size = faces.size(0);
    const auto num_faces = faces.size(1);
    const auto voxel_size = voxels.size(1);
    const int threads = 512;
    const dim3 blocks ((batch_size * num_faces - 1) / threads +1);

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "voxelize_sub2_cuda", ([&] {
      voxelize_sub2_kernel<scalar_t><<<blocks, threads>>>(
          faces.data<scalar_t>(),
          voxels.data<int32_t>(),
          batch_size,
          num_faces,
          voxel_size);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in voxelize_sub2_kernel: %s\n", hipGetErrorString(err));

    return {voxels};
}

std::vector<at::Tensor> voxelize_sub3_cuda(
        at::Tensor faces,
        at::Tensor voxels,
        at::Tensor visible) {

    const auto batch_size = voxels.size(0);
    const auto voxel_size = voxels.size(1);
    const int threads = 512;
    const dim3 blocks ((batch_size * voxel_size  * voxel_size  * voxel_size - 1) / threads +1);

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "voxelize_sub3_cuda", ([&] {
      voxelize_sub3_kernel<scalar_t><<<blocks, threads>>>(
          voxels.data<int32_t>(),
          visible.data<int32_t>(),
          batch_size,
          voxel_size);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in voxelize_sub3_kernel: %s\n", hipGetErrorString(err));

    return {voxels, visible};
}

std::vector<at::Tensor> voxelize_sub4_cuda(
        at::Tensor faces,
        at::Tensor voxels,
        at::Tensor visible) {

    const auto batch_size = voxels.size(0);
    const auto voxel_size = voxels.size(1);
    const int threads = 512;
    const dim3 blocks ((batch_size * voxel_size  * voxel_size  * voxel_size - 1) / threads +1);

    AT_DISPATCH_FLOATING_TYPES(faces.type(), "voxelize_sub4_cuda", ([&] {
      voxelize_sub4_kernel<scalar_t><<<blocks, threads>>>(
          voxels.data<int32_t>(),
          visible.data<int32_t>(),
          batch_size,
          voxel_size);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in voxelize_sub4_kernel: %s\n", hipGetErrorString(err));

    return {voxels, visible};
}