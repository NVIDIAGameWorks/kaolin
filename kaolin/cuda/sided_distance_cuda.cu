// Copyright (c) 2019-2020, NVIDIA CORPORATION. All rights reserved.

// Permission is hereby granted, free of charge, to any person obtaining
// a copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the Software
// is furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
// INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
// PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
// OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
// SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include <hip/hip_runtime.h>

#include "device_atomic_functions.h"


__global__ void SidedDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){

    const int batch=512;
    __shared__ float buf[batch*3];
    for (int i=blockIdx.x;i<b;i+=gridDim.x){
        for (int k2=0;k2<m;k2+=batch){
            int end_k=min(m,k2+batch)-k2;
            for (int j=threadIdx.x;j<end_k*3;j+=blockDim.x){
                buf[j]=xyz2[(i*m+k2)*3+j];
            }
            __syncthreads();
            for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
                float x1=xyz[(i*n+j)*3+0];
                float y1=xyz[(i*n+j)*3+1];
                float z1=xyz[(i*n+j)*3+2];
                int best_i=0;
                float best=0;
                int end_ka=end_k-(end_k&3);
                if (end_ka==batch){
                    for (int k=0;k<batch;k+=4){
                        {
                            float x2=buf[k*3+0]-x1;
                            float y2=buf[k*3+1]-y1;
                            float z2=buf[k*3+2]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (k==0 || d<best){
                                best=d;
                                best_i=k+k2;
                            }
                        }
                        {
                            float x2=buf[k*3+3]-x1;
                            float y2=buf[k*3+4]-y1;
                            float z2=buf[k*3+5]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (d<best){
                                best=d;
                                best_i=k+k2+1;
                            }
                        }
                        {
                            float x2=buf[k*3+6]-x1;
                            float y2=buf[k*3+7]-y1;
                            float z2=buf[k*3+8]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (d<best){
                                best=d;
                                best_i=k+k2+2;
                            }
                        }
                        {
                            float x2=buf[k*3+9]-x1;
                            float y2=buf[k*3+10]-y1;
                            float z2=buf[k*3+11]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (d<best){
                                best=d;
                                best_i=k+k2+3;
                            }
                        }
                    }
                }else{
                    for (int k=0;k<end_ka;k+=4){
                        {
                            float x2=buf[k*3+0]-x1;
                            float y2=buf[k*3+1]-y1;
                            float z2=buf[k*3+2]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (k==0 || d<best){
                                best=d;
                                best_i=k+k2;
                            }
                        }
                        {
                            float x2=buf[k*3+3]-x1;
                            float y2=buf[k*3+4]-y1;
                            float z2=buf[k*3+5]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (d<best){
                                best=d;
                                best_i=k+k2+1;
                            }
                        }
                        {
                            float x2=buf[k*3+6]-x1;
                            float y2=buf[k*3+7]-y1;
                            float z2=buf[k*3+8]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (d<best){
                                best=d;
                                best_i=k+k2+2;
                            }
                        }
                        {
                            float x2=buf[k*3+9]-x1;
                            float y2=buf[k*3+10]-y1;
                            float z2=buf[k*3+11]-z1;
                            float d=x2*x2+y2*y2+z2*z2;
                            if (d<best){
                                best=d;
                                best_i=k+k2+3;
                            }
                        }
                    }
                }
                for (int k=end_ka;k<end_k;k++){
                    float x2=buf[k*3+0]-x1;
                    float y2=buf[k*3+1]-y1;
                    float z2=buf[k*3+2]-z1;
                    float d=x2*x2+y2*y2+z2*z2;
                    if (k==0 || d<best){
                        best=d;
                        best_i=k+k2;
                    }
                }
                if (k2==0 || result[(i*n+j)]>best){
                    result[(i*n+j)]=best;
                    result_i[(i*n+j)]=best_i;
                }
            }
            __syncthreads();
        }
    }
}

void SidedDistanceKernelLauncher(
    const int b, const int n,
    const float* xyz,
    const int m,
    const float* xyz2,
    float* result,
    int* result_i)
{
    SidedDistanceKernel<<<dim3(32,16,1),512>>>( b, n, xyz, m, xyz2, result, result_i);
}


