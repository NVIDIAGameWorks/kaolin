#include "hip/hip_runtime.h"
// Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES.
// All rights reserved.

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//    http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#define CUB_NS_PREFIX namespace kaolin {
#define CUB_NS_POSTFIX }
#define CUB_NS_QUALIFIER ::kaolin::cub

#include <cub/device/device_scan.cuh>

#include <ATen/ATen.h>

#include "../../spc_math.h"
#include "../../utils.h"
#include "../../spc_utils.cuh"

#define THREADS_PER_BLOCK 64

namespace kaolin {

__global__ void scan_nodes_cuda_kernel(
    const uint num_bytes,
    const uint8_t *octree,
    uint *octrees_ptr) {
  uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if (tidx < num_bytes)
    octrees_ptr[tidx] = __popc(octree[tidx]);
}

int scan_octrees_cuda_impl(
    at::Tensor octrees,
    at::Tensor lengths,
    at::Tensor num_childrens_per_node,
    at::Tensor prefix_sum,
    at::Tensor pyramid) {
  int batch_size = lengths.size(0);
  // get tensor data pointers
  uint8_t* octrees_ptr = octrees.data_ptr<uint8_t>();
  uint* num_childrens_per_node_ptr = reinterpret_cast<uint*>(num_childrens_per_node.data_ptr<int>());
  uint* prefix_sum_ptr = reinterpret_cast<uint*>(prefix_sum.data_ptr<int>());
  int* pyramid_ptr = pyramid.data_ptr<int>();
  
  void* temp_storage_ptr = NULL;
  uint64_t temp_storage_bytes = get_cub_storage_bytes(
        temp_storage_ptr, num_childrens_per_node_ptr, prefix_sum_ptr, num_childrens_per_node.size(0) + 1);
  at::Tensor temp_storage = at::zeros({(int64_t) temp_storage_bytes },
                                      octrees.options().dtype(at::kByte));
  temp_storage_ptr = (void*) temp_storage.data_ptr<uint8_t>();

  // TODO: document better
  uint* EX0 = prefix_sum_ptr;
  uint8_t* O0 = octrees_ptr;
  int* h0 = pyramid_ptr;
  int level;

  for (int batch = 0; batch < batch_size; batch++) {
    uint8_t*  O = O0;
    uint*   S = EX0 + 1;
    uint  osize = lengths[batch].item<int>();

    // compute exclusive sum 1 element beyond end of list to get inclusive sum starting at prefix_sum_ptr+1
    scan_nodes_cuda_kernel<<< (osize + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(
        osize, O0, num_childrens_per_node_ptr);
    CubDebugExit(hipcub::DeviceScan::ExclusiveSum(
        temp_storage_ptr, temp_storage_bytes, num_childrens_per_node_ptr,
        EX0, osize + 1)); // carful with the +1

    int* Pmid = h0;
    int* PmidSum = h0 + KAOLIN_SPC_MAX_LEVELS + 2;

    int Lsize = 1;
    uint currSum, prevSum = 0;

    uint sum = Pmid[0] = Lsize;
    PmidSum[0] = 0;
    PmidSum[1] = Lsize;

    level = 0;
    while (sum <= osize) {
      O += Lsize;
      S += Lsize;

      hipMemcpy(&currSum, EX0 + prevSum + 1, sizeof(uint), hipMemcpyDeviceToHost);
      CUDA_CHECK(hipGetLastError());

      Lsize = currSum - prevSum;
      prevSum = currSum;

      Pmid[++level] = Lsize;
      sum += Lsize;
      PmidSum[level + 1] = sum;
    }

    O0 += osize;
    EX0 += (osize + 1);
    h0 += 2 * (KAOLIN_SPC_MAX_LEVELS + 2);
  }
  CUDA_CHECK(hipGetLastError());

  return level;
}

}  // namespace kaolin
