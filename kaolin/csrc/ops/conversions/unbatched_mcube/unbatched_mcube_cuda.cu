#include "hip/hip_runtime.h"
// Copyright (c) 2019-2020, NVIDIA CORPORATION. All rights reserved.

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//    http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#define CUB_NS_PREFIX namespace kaolin {
#define CUB_NS_POSTFIX }
#define CUB_NS_QUALIFIER ::kaolin::cub

#include <stdio.h>
#include <string.h>

#include "tables.h"
#include "hip/hip_vector_types.h"

#include <ATen/ATen.h>

#include <hipcub/hipcub.hpp>

namespace kaolin {

// textures containing look-up tables
texture<uint, 1, hipReadModeElementType> triTex;
texture<uint, 1, hipReadModeElementType> numUniqueVertsTex;
texture<uint, 1, hipReadModeElementType> numTrianglesTex;
texture<uint, 1, hipReadModeElementType> numPartialVertsTex;
texture<uint, 1, hipReadModeElementType> vertsOrderTex;

void allocateTextures(at::Tensor d_triTable, at::Tensor d_numUniqueVertsTable, 
                      at::Tensor d_numTrianglesTable, at::Tensor d_numPartialVertsTable,
                      at::Tensor d_vertsOrderTable)
{
  // TODO: rename allocateTextures
  // TODO: check if texture is already binded.
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

  hipMemcpy((void *) d_triTable.data_ptr<int>(), (void *)triTable, 256*16*sizeof(int), hipMemcpyHostToDevice);
  hipBindTexture(0, triTex, d_triTable.data_ptr<int>(), channelDesc);

  hipMemcpy((void *) d_numUniqueVertsTable.data_ptr<int>(), (void *)numUniqueVertsTable, 256*sizeof(int), hipMemcpyHostToDevice);
  hipBindTexture(0, numUniqueVertsTex, d_numUniqueVertsTable.data_ptr<int>(), channelDesc);

  hipMemcpy((void *) d_numTrianglesTable.data_ptr<int>(), (void *)numTrianglesTable, 256*sizeof(int), hipMemcpyHostToDevice);
  hipBindTexture(0, numTrianglesTex, d_numTrianglesTable.data_ptr<int>(), channelDesc);

  hipMemcpy((void *) d_numPartialVertsTable.data_ptr<int>(), (void *)numPartialVertsTable, 256*sizeof(int), hipMemcpyHostToDevice);
  hipBindTexture(0, numPartialVertsTex, d_numPartialVertsTable.data_ptr<int>(), channelDesc);

  hipMemcpy((void *) d_vertsOrderTable.data_ptr<int>(), (void *)vertsOrderTable, 256*3*sizeof(int), hipMemcpyHostToDevice);
  hipBindTexture(0, vertsOrderTex, d_vertsOrderTable.data_ptr<int>(), channelDesc);
}

// sample volume data set at a point
__device__
float sampleVolume(float* data, int3 p, int3 gridSize)
{
  p.x = min(p.x, gridSize.x - 1);
  p.y = min(p.y, gridSize.y - 1);
  p.z = min(p.z, gridSize.z - 1);
  int i = (p.z*gridSize.x*gridSize.y) + (p.y*gridSize.x) + p.x;

  return data[i];
}

// compute position in 3d grid from 1d index
__device__
int3 calcGridPos(int i, int3 gridSize)
{
  int3 gridPos;
  gridPos.x = i % gridSize.x;
  gridPos.y = (i / gridSize.x) % gridSize.y;
  gridPos.z = (i / gridSize.x / gridSize.y) % gridSize.z;
  return gridPos;
}

__global__ void
classifyVoxel(int *voxelOccupied, int *voxelTriangles, int *voxelPartialVerts,
              int *voxelVertsOrder,
              float* volume, int3 gridSize, int numVoxels,
              float3 voxelSize, float isoValue)
{
  int blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  int i = __mul24(blockId, blockDim.x) + threadIdx.x;

  int3 gridPos = calcGridPos(i, gridSize);

  // read field values at neighbouring grid vertices// find the target voxel index which is responsible to generate the vertex
  float field[8];
  field[0] = sampleVolume(volume, gridPos, gridSize);
  field[1] = sampleVolume(volume, gridPos + make_int3(1, 0, 0), gridSize);
  field[2] = sampleVolume(volume, gridPos + make_int3(1, 1, 0), gridSize);
  field[3] = sampleVolume(volume, gridPos + make_int3(0, 1, 0), gridSize);
  field[4] = sampleVolume(volume, gridPos + make_int3(0, 0, 1), gridSize);
  field[5] = sampleVolume(volume, gridPos + make_int3(1, 0, 1), gridSize);
  field[6] = sampleVolume(volume, gridPos + make_int3(1, 1, 1), gridSize);
  field[7] = sampleVolume(volume, gridPos + make_int3(0, 1, 1), gridSize);

  // calculate flag indicating if each vertex is inside or outside isosurface
  int cubeindex;
  cubeindex =  int(field[0] < isoValue);
  cubeindex += int(field[1] < isoValue)*2;
  cubeindex += int(field[2] < isoValue)*4;
  cubeindex += int(field[3] < isoValue)*8;
  cubeindex += int(field[4] < isoValue)*16;
  cubeindex += int(field[5] < isoValue)*32;
  cubeindex += int(field[6] < isoValue)*64;
  cubeindex += int(field[7] < isoValue)*128;

  // read number of vertices from texture for half cube
  int numVerts = tex1Dfetch(numUniqueVertsTex, cubeindex);
  int numPartialVerts = tex1Dfetch(numPartialVertsTex, cubeindex);
  int numTriangles = tex1Dfetch(numTrianglesTex, cubeindex);

  int vertsOrder1 = tex1Dfetch(vertsOrderTex, cubeindex*3);
  int vertsOrder2 = tex1Dfetch(vertsOrderTex, cubeindex*3 + 1);
  int vertsOrder3 = tex1Dfetch(vertsOrderTex, cubeindex*3 + 2);

  if (i < numVoxels)
  {
    voxelPartialVerts[i] = numPartialVerts;
    voxelOccupied[i] = (numVerts > 0);
    voxelTriangles[i] = numTriangles;
    
    voxelVertsOrder[i*3] = vertsOrder1;
    voxelVertsOrder[i*3 + 1] = vertsOrder2;
    voxelVertsOrder[i*3 + 2] = vertsOrder3;
  }
}

void launch_classifyVoxel(at::Tensor voxelOccupied, at::Tensor voxelTriangles, at::Tensor voxelPartialVerts,
                          at::Tensor voxelVertsOrder,
                          int3 gridSize, int numVoxels, at::Tensor voxelgrid,
                          float3 voxelSize, float isoValue)
{   
  int threads;
  dim3 grid(1, 1, 1);

  // For smaller voxelgrid
  if (numVoxels < 128) {
    threads = numVoxels;
  } else {
    threads = 128;
    grid.x = numVoxels / threads;
  }

  // get around maximum grid size of 65535 in each dimension
  if (grid.x > 65535)
  {
    grid.y = grid.x / 32768;
    grid.x = 32768;
  }
  // calculate number of vertices need per voxel
  classifyVoxel<<<grid, threads>>>(voxelOccupied.data_ptr<int>(), 
                                   voxelTriangles.data_ptr<int>(), voxelPartialVerts.data_ptr<int>(),
                                   voxelVertsOrder.data_ptr<int>(),
                                   voxelgrid.data_ptr<float>(), gridSize,
                                   numVoxels, voxelSize, isoValue);
}

// compact voxel array
__global__ void
compactVoxels(int *compactedVoxelArray, int *voxelOccupied, int *voxelOccupiedScan, int numVoxels)
{
  int blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  int i = __mul24(blockId, blockDim.x) + threadIdx.x;

  if (voxelOccupied[i] && (i < numVoxels))
  {
    compactedVoxelArray[voxelOccupiedScan[i]] = i;
  }
}

void launch_compactVoxels(at::Tensor compactedVoxelArray, at::Tensor voxelOccupied, at::Tensor voxelOccupiedScan, int numVoxels)
{
  int threads;
  dim3 grid(1, 1, 1);

  // For smaller voxelgrid
  if (numVoxels < 128) {
    threads = numVoxels;
  } else {
    threads = 128;
    grid.x = numVoxels / threads;
  }

  // get around maximum grid size of 65535 in each dimension
  if (grid.x > 65535)
  {
    grid.y = grid.x / 32768;
    grid.x = 32768;
  }

  compactVoxels<<<grid, threads>>>(compactedVoxelArray.data_ptr<int>(), voxelOccupied.data_ptr<int>(),
                                   voxelOccupiedScan.data_ptr<int>(), numVoxels);
}

// compute interpolated vertex along an edge
__device__
float3 vertexInterp(float isolevel, float3 p0, float3 p1, float f0, float f1)
{
  float t = (isolevel - f0) / (f1 - f0);
  return lerp(p0, p1, t);
}

// find the target voxel index which is responsible to generate the vertex
__device__
int find_target_voxel(int3 gridSize, int face_idx, int current_voxel)
{
  int target_voxel_idx;
  // x-axis increase -> to the right
  // y-axis increase -> to the top
  // z-axis increase -> to the back
  switch(face_idx) {
    case 0:  // looking for vertices in bot-front voxel
      target_voxel_idx = current_voxel - gridSize.x - gridSize.x*gridSize.y;
      break;
  
    case 1:  // looking for vertices in right-front voxel
      target_voxel_idx = current_voxel + 1 - gridSize.x*gridSize.y;
      break;

    case 2:  // looking for vertices in front voxel
      target_voxel_idx = current_voxel - gridSize.x*gridSize.y;
      break;

    case 3: // looking for vertices in front voxel
      target_voxel_idx = current_voxel - gridSize.x*gridSize.y;
      break;

    case 4: // looking for vertices in bot voxel
      target_voxel_idx = current_voxel - gridSize.x;
      break;

    case 5: // looking for vertices in right voxel
      target_voxel_idx = current_voxel + 1;
      break;
  
    case 6:  // looking for vertices in current voxel
      target_voxel_idx = current_voxel;
      break;

    case 7:  // looking for vertices in current voxel
      target_voxel_idx = current_voxel;
      break;

    case 8: // looking for vertices in bot voxel
      target_voxel_idx = current_voxel - gridSize.x;
      break;

    case 9: // looking for vertices in right-bot voxel
      target_voxel_idx = current_voxel + 1 - gridSize.x;
      break;

    case 10: // looking for vertices in right voxel
      target_voxel_idx = current_voxel + 1;
      break;
  
    case 11: // looking for vertices in current voxel
      target_voxel_idx = current_voxel;
      break;

    default:
      target_voxel_idx = current_voxel;
      break;
    }
  return target_voxel_idx;
}

// find the offset, given the vertex is on what edge
__device__
int find_offset(int face_idx, int voxel_index, int* voxelVertsOrder)
{
  int offset;
  int corresponding_edge;  // corresponding edge number in current voxel_index

  switch(face_idx) {
    case 0:  // looking for vertices in bot-front voxel
      corresponding_edge = 6; // corresponds to edge 6
      break;
  
    case 1:  // looking for vertices in right-front voxel
      corresponding_edge = 7; // corresponds to edge 7
      break;

    case 2:  // looking for vertices in front voxel
      corresponding_edge = 6; // corresponds to edge 6
      break;

    case 3: // looking for vertices in front voxel
      corresponding_edge = 7; // corresponds to edge 7
      break;

    case 4: // looking for vertices in bot voxel
      corresponding_edge = 6; // corresponds to edge 6
      break;

    case 5: // looking for vertices in right voxel
      corresponding_edge = 7; // corresponds to edge 7
      break;
  
    case 6:  // looking for vertices in current voxel
      corresponding_edge = 6;
      break;

    case 7:  // looking for vertices in current voxel
      corresponding_edge = 7;
      break;

    case 8: // looking for vertices in bot voxel
      corresponding_edge = 11; // corresponds to edge 11
      break;

    case 9: // looking for vertices in right-bot voxel
      corresponding_edge = 11; // corresponds to edge 11
      break;

    case 10: // looking for vertices in right voxel
      corresponding_edge = 11; // corresponds to edge 11
      break;
  
    case 11: // looking for vertices in current voxel
      corresponding_edge = 11;
      break;

    default:
      corresponding_edge = face_idx;
      break;
  }

  int verts_order_1 = voxelVertsOrder[voxel_index * 3];
  int verts_order_2 = voxelVertsOrder[voxel_index * 3 + 1];
  int verts_order_3 = voxelVertsOrder[voxel_index * 3 + 2];

  if (verts_order_1 == 255 && verts_order_2 == 255 && verts_order_3 == 255) {
    return 0;
  }

  if (corresponding_edge == verts_order_1) {
    offset = 0;
  } else if (corresponding_edge == verts_order_2) {
    offset = 1;
  } else if (corresponding_edge == verts_order_3) {
    offset = 2;
  }

  return offset;
}


// version that calculates flat surface normal for each triangle
__global__ void
generateTriangles2(float *pos, int *faces, int *compactedVoxelArray,
                   int *numTrianglesScanned, int *numPartialVertsScanned, int *numPartialVerts,
                   int *voxelVertsOrder,
                   float* volume, int3 gridSize,
                   float3 voxelSize, float isoValue, int activeVoxels, int maxVerts)
{
  int blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  int grid_index = __mul24(blockId, blockDim.x) + threadIdx.x;

  if (grid_index > activeVoxels - 1)
  {
    grid_index = activeVoxels - 1;
  }

  int voxel = compactedVoxelArray[grid_index];

  // compute position in 3d grid
  int3 gridPos = calcGridPos(voxel, gridSize);

  float3 p;

  p.x = gridPos.x;
  p.y = gridPos.y;
  p.z = gridPos.z;

  // calculate unnormalized cell vertex positions
  float3 v[8];
  v[0] = p;
  v[1] = p + make_float3(1, 0, 0);
  v[2] = p + make_float3(1, 1, 0);
  v[3] = p + make_float3(0, 1, 0);
  v[4] = p + make_float3(0, 0, 1);
  v[5] = p + make_float3(1, 0, 1);
  v[6] = p + make_float3(1, 1, 1);
  v[7] = p + make_float3(0, 1, 1);

  float field[8];
  field[0] = sampleVolume(volume, gridPos, gridSize);
  field[1] = sampleVolume(volume, gridPos + make_int3(1, 0, 0), gridSize);
  field[2] = sampleVolume(volume, gridPos + make_int3(1, 1, 0), gridSize);
  field[3] = sampleVolume(volume, gridPos + make_int3(0, 1, 0), gridSize);
  field[4] = sampleVolume(volume, gridPos + make_int3(0, 0, 1), gridSize);
  field[5] = sampleVolume(volume, gridPos + make_int3(1, 0, 1), gridSize);
  field[6] = sampleVolume(volume, gridPos + make_int3(1, 1, 1), gridSize);
  field[7] = sampleVolume(volume, gridPos + make_int3(0, 1, 1), gridSize);

  // recalculate flag
  int cubeindex;
  cubeindex =  int(field[0] < isoValue);
  cubeindex += int(field[1] < isoValue)*2;
  cubeindex += int(field[2] < isoValue)*4;
  cubeindex += int(field[3] < isoValue)*8;
  cubeindex += int(field[4] < isoValue)*16;
  cubeindex += int(field[5] < isoValue)*32;
  cubeindex += int(field[6] < isoValue)*64;
  cubeindex += int(field[7] < isoValue)*128;

  // find the vertices where the surface intersects the cube    
  // use shared memory to avoid using local
  __shared__ float3 vertlist[12*NTHREADS];

  vertlist[threadIdx.x] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
  vertlist[NTHREADS+threadIdx.x] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
  vertlist[(NTHREADS*2)+threadIdx.x] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
  vertlist[(NTHREADS*3)+threadIdx.x] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);
  vertlist[(NTHREADS*4)+threadIdx.x] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
  vertlist[(NTHREADS*5)+threadIdx.x] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
  vertlist[(NTHREADS*6)+threadIdx.x] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
  vertlist[(NTHREADS*7)+threadIdx.x] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);
  vertlist[(NTHREADS*8)+threadIdx.x] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
  vertlist[(NTHREADS*9)+threadIdx.x] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
  vertlist[(NTHREADS*10)+threadIdx.x] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
  vertlist[(NTHREADS*11)+threadIdx.x] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);

  __syncthreads();

  int added_vertx_count = 0;
  for (int i=0; i<3; i++) { // maximum 3 newly added vertices for a voxel

    float3 *v[1];

    uint edge = tex1Dfetch(vertsOrderTex, (cubeindex*3) + i);

    if (edge == 255) {
        break;
    }

    // Only add the top-left-back vertices of the cube to the vertices' list
    // Meaning only vertices on the edge 6, 7, 11
    int index = numPartialVertsScanned[voxel] + added_vertx_count;
    added_vertx_count++;

    v[0] = &vertlist[(edge*NTHREADS)+threadIdx.x];

    // Add the vertex in reverse order to keep the original pose.
    if (index < (maxVerts - 3)) {
        pos[index * 3] = (v[0]) -> z;
        pos[index * 3 + 1] = (v[0]) -> y;
        pos[index * 3 + 2] = (v[0]) -> x;
    }
  }

    // Add triangles
  for (int j=0; j<16; j+=3) {
    uint face_idx1 = tex1Dfetch(triTex, cubeindex*16 + j);

    if (face_idx1 == 255) {
      break;
    }

    uint face_idx2 = tex1Dfetch(triTex, cubeindex*16 + j + 1);
    uint face_idx3 = tex1Dfetch(triTex, cubeindex*16 + j + 2);

    int num_prev_verts;
    int num_prev_triangles;

    int target_voxel_idx1 = find_target_voxel(gridSize, face_idx1, voxel);
    int target_voxel_idx2 = find_target_voxel(gridSize, face_idx2, voxel);
    int target_voxel_idx3 = find_target_voxel(gridSize, face_idx3, voxel);

    int offset1 = find_offset(face_idx1, target_voxel_idx1, voxelVertsOrder);
    int offset2 = find_offset(face_idx2, target_voxel_idx2, voxelVertsOrder); 
    int offset3 = find_offset(face_idx3, target_voxel_idx3, voxelVertsOrder);

    // Add the faces in reverse order to ensure that original pose is unchanged
    // handle first vertex
    num_prev_verts = numPartialVertsScanned[target_voxel_idx1];
    num_prev_triangles = numTrianglesScanned[voxel];

    faces[num_prev_triangles * 3 + j + 2] = num_prev_verts + offset1;

    // handle second vertex
    num_prev_verts = numPartialVertsScanned[target_voxel_idx2];
    num_prev_triangles = numTrianglesScanned[voxel];

    faces[num_prev_triangles * 3 + j + 1] = num_prev_verts + offset2;

    // handle last vertex
    num_prev_verts = numPartialVertsScanned[target_voxel_idx3];
    num_prev_triangles = numTrianglesScanned[voxel];

    faces[num_prev_triangles * 3 + j] = num_prev_verts + offset3;
  }
}

void launch_generateTriangles2(at::Tensor pos, at::Tensor faces, at::Tensor compactedVoxelArray,
                               at::Tensor numTrianglesScanned, at::Tensor numPartialVertsScanned,
                               at::Tensor numPartialVerts, at::Tensor voxelVertsOrder,
                               int3 gridSize, at::Tensor voxelgrid,
                               float3 voxelSize, float isoValue, int activeVoxels, int maxVerts)
{
  dim3 grid2((int) ceil(activeVoxels/ (float) NTHREADS), 1, 1);

  while (grid2.x > 65535) {
    grid2.x/=2;
    grid2.y*=2;
  }

  generateTriangles2<<<grid2, NTHREADS>>>(pos.data_ptr<float>(), faces.data_ptr<int>(),
                                          compactedVoxelArray.data_ptr<int>(), numTrianglesScanned.data_ptr<int>(),
                                          numPartialVertsScanned.data_ptr<int>(), numPartialVerts.data_ptr<int>(),
                                          voxelVertsOrder.data_ptr<int>(),
                                          voxelgrid.data_ptr<float>(), gridSize,
                                          voxelSize, isoValue, activeVoxels,
                                          maxVerts);
}

void CubScanWrapper(at::Tensor output, at::Tensor input, int numElements) {
  int *d_in = input.data_ptr<int>();
  int *d_out = output.data_ptr<int>();

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, numElements);

  // Allocate temporary storage
  at::Tensor temp_storage = at::zeros({(int) temp_storage_bytes}, input.options());
  d_temp_storage = temp_storage.data_ptr<int>();

  // Run exclusive prefix sum
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, numElements);
}
// at::Tensor used to store tables
at::Tensor d_triTable;
at::Tensor d_numUniqueVertsTable;
at::Tensor d_numTrianglesTable;
at::Tensor d_numPartialVertsTable;
at::Tensor d_vertsOrderTable;

void
computeIsosurface(int3 gridSize, int3 gridSizeLog2, float isoValue,
                  int *activeVoxels, int *totalVerts, int *totalTriangles, int *totalPartialVerts,
                  int numVoxels, float3 voxelSize, int maxVerts, int maxFaces,
                  at::Tensor voxelgrid, at::Tensor d_pos, at::Tensor d_faces,
                  at::Tensor d_voxelPartialVerts,
                  at::Tensor d_voxelTriangles,
                  at::Tensor d_voxelOccupied,
                  at::Tensor d_compVoxelArray,
                  at::Tensor d_voxelVertsOrder) {
  // calculate number of vertices and triangles need per voxel
  launch_classifyVoxel(d_voxelOccupied,
                       d_voxelTriangles, d_voxelPartialVerts,
                       d_voxelVertsOrder,
                       gridSize, numVoxels, voxelgrid,
                       voxelSize, isoValue);
  
  at::Tensor d_voxelOccupiedScan = at::zeros({numVoxels}, voxelgrid.options().dtype(at::kInt));
  CubScanWrapper(d_voxelOccupiedScan, d_voxelOccupied, numVoxels);
  // read back values to calculate total number of non-empty voxels
  // since we are using an exclusive scan, the total is the last value of
  // the scan result plus the last value in the input array
  {
    int lastElement, lastScanElement;
    hipMemcpy((void *) &lastElement,
               (void *)(d_voxelOccupied.data_ptr<int>() + numVoxels - 1),
               sizeof(int), hipMemcpyDeviceToHost);
  
    hipMemcpy((void *) &lastScanElement,
               (void *)(d_voxelOccupiedScan.data_ptr<int>() + numVoxels - 1),
               sizeof(int), hipMemcpyDeviceToHost);
  
    *activeVoxels = lastElement + lastScanElement;
  }

  if (activeVoxels==0)
  {
    // return if there are no full voxels
    *totalVerts = 0;
    return;
  }

  // compact voxel index array
  launch_compactVoxels(d_compVoxelArray, d_voxelOccupied, d_voxelOccupiedScan, numVoxels);

  //scan voxel triangle count array
  at::Tensor d_voxelTrianglesScan = at::zeros({numVoxels}, voxelgrid.options().dtype(at::kInt));
  CubScanWrapper(d_voxelTrianglesScan, d_voxelTriangles, numVoxels);

  //scan partial vertex count array
  at::Tensor d_voxelPartialVertsScan = at::zeros({numVoxels}, voxelgrid.options().dtype(at::kInt));
  CubScanWrapper(d_voxelPartialVertsScan, d_voxelPartialVerts, numVoxels);

  // readback total number of triangles
  {
    int lastElement, lastScanElement;
    hipMemcpy((void *) &lastElement,
               (void *)(d_voxelTriangles.data_ptr<int>() + numVoxels-1),
               sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy((void *) &lastScanElement,
               (void *)(d_voxelTrianglesScan.data_ptr<int>() + numVoxels-1),
               sizeof(int), hipMemcpyDeviceToHost);
    *totalTriangles = lastElement + lastScanElement;
  }

  // readback total number of partial verts
  {
    int lastElement, lastScanElement;
    hipMemcpy((void *) &lastElement,
               (void *)(d_voxelPartialVerts.data_ptr<int>() + numVoxels-1),
               sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy((void *) &lastScanElement,
               (void *)(d_voxelPartialVertsScan.data_ptr<int>() + numVoxels-1),
               sizeof(int), hipMemcpyDeviceToHost);
    *totalPartialVerts = lastElement + lastScanElement;
  }

  launch_generateTriangles2(d_pos,
                            d_faces,
                            d_compVoxelArray,
                            d_voxelTrianglesScan,
                            d_voxelPartialVertsScan,
                            d_voxelPartialVerts,
                            d_voxelVertsOrder,
                            gridSize, voxelgrid,
                            voxelSize, isoValue, *activeVoxels,
                            maxVerts);
}

std::vector<at::Tensor> unbatched_mcube_forward_cuda_kernel_launcher(const at::Tensor voxelgrid, float iso_value) {
  int3 gridSizeLog2;
  int3 gridSize;  // height, width, depth of voxelgrid

  float3 voxelSize;
  int numVoxels         = 0;  // numbel of total voxels of the input voxelgrid
  int maxVerts          = 0;  // maximum number of vertices of the output mesh could have
  int maxFaces          = 0;  // maximum number of faces of the output mesh could have
  int activeVoxels      = 0;  // number of total voxels that have vertices
  int totalVerts        = 0;  // number of actual generated vertices of the output mesh
  int totalTriangles    = 0;  // number of actual generated triangles(faces) of the output mesh
  int totalPartialVerts = 0;  // number of vertices generated by all voxel, but only count on three edges.

  float isoValue = 0.5f;
  isoValue = iso_value;

  int i = voxelgrid.size(0);
  int j = voxelgrid.size(1);
  int k = voxelgrid.size(2);

  gridSizeLog2.x = (int) log2(i);
  gridSizeLog2.y = (int) log2(j);
  gridSizeLog2.z = (int) log2(k);

  gridSize = make_int3(i, j, k);

  numVoxels = gridSize.x*gridSize.y*gridSize.z;
  voxelSize = make_float3(2.0f / gridSize.x, 2.0f / gridSize.y, 2.0f / gridSize.z);
  maxVerts = gridSize.x*gridSize.y*100;
  maxFaces = numVoxels * 6;

  // initialize tensors
  auto int_options = voxelgrid.options().dtype(at::kInt);

  at::Tensor d_pos = at::zeros({maxVerts, 3}, voxelgrid.options().dtype(at::kFloat)); // tensor to store output vertices

  at::Tensor d_faces = at::zeros({maxFaces, 3}, int_options); // tensor to store output faces

  at::Tensor d_voxelPartialVerts = at::zeros({numVoxels}, int_options); // tensor to measure how many vertices a voxel will generate, only count on three edges.
  at::Tensor d_voxelTriangles = at::zeros({numVoxels}, int_options); // tensor to measure how many trianlges a voxel will generate
  at::Tensor d_voxelOccupied = at::zeros({numVoxels}, int_options); // binary tensor to indicate whether the voxel will generate any vertices or not
  at::Tensor d_compVoxelArray = at::zeros({numVoxels}, int_options); // compact representation of d_voxelOccupiedScan

  at::Tensor d_voxelVertsOrder = at::zeros({numVoxels, 3}, int_options); // tensor to store the order of added verts for each voxel

  // initialize static pointers
  if (!d_triTable.defined()) {
    d_triTable = at::zeros({256, 16}, int_options);
    d_numUniqueVertsTable = at::zeros({256}, int_options);
    d_numTrianglesTable = at::zeros({256}, int_options);
    d_numPartialVertsTable = at::zeros({256}, int_options);
    d_vertsOrderTable = at::zeros({256, 3}, int_options);

    // allocate table textures after we initialize everything.
    allocateTextures(d_triTable, d_numUniqueVertsTable, d_numTrianglesTable, d_numPartialVertsTable, d_vertsOrderTable);
  }

  computeIsosurface(gridSize, gridSizeLog2, isoValue,
                    &activeVoxels, &totalVerts, &totalTriangles, &totalPartialVerts,
                    numVoxels, voxelSize, maxVerts, maxFaces,
                    voxelgrid, d_pos, d_faces,
                    d_voxelPartialVerts,
                    d_voxelTriangles,
                    d_voxelOccupied, d_compVoxelArray, d_voxelVertsOrder);

  std::vector<at::Tensor> result;

  result.push_back(d_pos.index({at::indexing::Slice(at::indexing::None, totalPartialVerts)}));
  result.push_back(d_faces.index({at::indexing::Slice(at::indexing::None, totalTriangles)}));

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();

  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }

  return result;
}

}  // namespace kaolin
